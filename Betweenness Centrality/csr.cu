#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>

__global__ void betweennessCentrality(int *row_ptr, int *col_ind, float *bc, int num_nodes) {
    int node = blockIdx.x * blockDim.x + threadIdx.x;
    if (node >= num_nodes) return;
    
    float centrality = 0.0f;
    int row_start = row_ptr[node];
    int row_end = row_ptr[node + 1];
    
    for (int i = row_start; i < row_end; i++) {
        int neighbor = col_ind[i];
        centrality += 1.0f; 
    }
    
    bc[node] = centrality;
}


void computeBetweennessCentrality(int *h_row_ptr, int *h_col_ind, int *h_adj_list, int *h_offset, float *h_bc, int num_nodes, bool use_csr) {
    int *d_row_ptr, *d_col_ind, *d_adj_list, *d_offset;
    float *d_bc;
    hipMalloc(&d_bc, num_nodes * sizeof(float));
    hipMemset(d_bc, 0, num_nodes * sizeof(float));
    
    if (use_csr) {
        hipMalloc(&d_row_ptr, (num_nodes + 1) * sizeof(int));
        hipMalloc(&d_col_ind, h_row_ptr[num_nodes] * sizeof(int));
        hipMemcpy(d_row_ptr, h_row_ptr, (num_nodes + 1) * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_col_ind, h_col_ind, h_row_ptr[num_nodes] * sizeof(int), hipMemcpyHostToDevice);
        
        betweennessCentrality<<<num_nodes, 256>>>(d_row_ptr, d_col_ind, d_bc, num_nodes);
        
        hipFree(d_row_ptr);
        hipFree(d_col_ind);

    
    hipMemcpy(h_bc, d_bc, num_nodes * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_bc);
}

int main() {
    int num_nodes = 5;
    int h_row_ptr[] = {0, 2, 5, 7, 9, 10};
    int h_col_ind[] = {1, 4, 0, 2, 4, 1, 3, 2, 4, 3};
    int h_adj_list[] = {1, 4, 0, 2, 4, 1, 3, 2, 4, 3};
    int h_offset[] = {0, 2, 5, 7, 9, 10};
    float h_bc[5] = {0};
    
    computeBetweennessCentrality(h_row_ptr, h_col_ind, h_adj_list, h_offset, h_bc, num_nodes, true);
    for (int i = 0; i < num_nodes; i++) {
        printf("Node %d -> Betweenness Centrality %f\n", i, h_bc[i]);
    }
    

    
    return 0;
}
