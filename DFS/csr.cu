#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>

__global__ void dfs(int *row_ptr, int *col_ind, int *visited, int *stack, int *stack_size, int num_nodes) {
    int node = blockIdx.x * blockDim.x + threadIdx.x;
    if (node >= num_nodes || visited[node]) return;
    
    visited[node] = 1;
    int row_start = row_ptr[node];
    int row_end = row_ptr[node + 1];
    
    for (int i = row_start; i < row_end; i++) {
        int neighbor = col_ind[i];
        if (!visited[neighbor]) {
            int index = atomicAdd(stack_size, 1);
            stack[index] = neighbor;
        }
    }
}


void computeDFS(int *h_row_ptr, int *h_col_ind, int *h_adj_list, int *h_offset, int num_nodes) {
    int *d_row_ptr, *d_col_ind, *d_adj_list, *d_offset, *d_visited, *d_stack, *d_stack_size;
    int h_stack_size = 0;
    int h_visited[num_nodes] = {0};
    int h_stack[num_nodes];
    
    hipMalloc(&d_visited, num_nodes * sizeof(int));
    hipMalloc(&d_stack, num_nodes * sizeof(int));
    hipMalloc(&d_stack_size, sizeof(int));
    hipMemcpy(d_visited, h_visited, num_nodes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_stack_size, &h_stack_size, sizeof(int), hipMemcpyHostToDevice);
    
        hipMalloc(&d_row_ptr, (num_nodes + 1) * sizeof(int));
        hipMalloc(&d_col_ind, h_row_ptr[num_nodes] * sizeof(int));
        hipMemcpy(d_row_ptr, h_row_ptr, (num_nodes + 1) * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_col_ind, h_col_ind, h_row_ptr[num_nodes] * sizeof(int), hipMemcpyHostToDevice);
        
        dfs<<<(num_nodes + 255) / 256, 256>>>(d_row_ptr, d_col_ind, d_visited, d_stack, d_stack_size, num_nodes);
        
        hipFree(d_row_ptr);
        hipFree(d_col_ind);

    
    hipMemcpy(h_visited, d_visited, num_nodes * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_visited);
    hipFree(d_stack);
    hipFree(d_stack_size);
    
    printf("DFS Traversal Result:\n");
    for (int i = 0; i < num_nodes; i++) {
        if (h_visited[i]) printf("Node %d is visited\n", i);
    }
}

int main() {
    int num_nodes = 5;
    int h_row_ptr[] = {0, 2, 5, 7, 9, 10};
    int h_col_ind[] = {1, 4, 0, 2, 4, 1, 3, 2, 4, 3};
    int h_adj_list[] = {1, 4, 0, 2, 4, 1, 3, 2, 4, 3};
    int h_offset[] = {0, 2, 5, 7, 9, 10};
    
    computeDFS(h_row_ptr, h_col_ind, h_adj_list, h_offset, num_nodes);
    
    
    return 0;
}
