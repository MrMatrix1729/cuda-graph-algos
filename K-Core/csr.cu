#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>

__global__ void kCore(int *row_ptr, int *col_ind, int *degrees, int k, int num_nodes) {
    int node = blockIdx.x * blockDim.x + threadIdx.x;
    if (node >= num_nodes) return;
    
    if (degrees[node] < k) {
        degrees[node] = 0;
        int row_start = row_ptr[node];
        int row_end = row_ptr[node + 1];
        for (int i = row_start; i < row_end; i++) {
            atomicSub(&degrees[col_ind[i]], 1);
        }
    }
}

void computeKCore(int *h_row_ptr, int *h_col_ind, int *h_adj_list, int *h_offset, int *h_degrees, int num_nodes, int k) {
    int *d_row_ptr, *d_col_ind, *d_adj_list, *d_offset, *d_degrees;
    hipMalloc(&d_degrees, num_nodes * sizeof(int));
    hipMemcpy(d_degrees, h_degrees, num_nodes * sizeof(int), hipMemcpyHostToDevice);
    
        hipMalloc(&d_row_ptr, (num_nodes + 1) * sizeof(int));
        hipMalloc(&d_col_ind, h_row_ptr[num_nodes] * sizeof(int));
        hipMemcpy(d_row_ptr, h_row_ptr, (num_nodes + 1) * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_col_ind, h_col_ind, h_row_ptr[num_nodes] * sizeof(int), hipMemcpyHostToDevice);
        
        kCore<<<num_nodes, 256>>>(d_row_ptr, d_col_ind, d_degrees, k, num_nodes);
        
        hipFree(d_row_ptr);
        hipFree(d_col_ind);
    
    
    hipMemcpy(h_degrees, d_degrees, num_nodes * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_degrees);
}

int main() {
    int num_nodes = 5;
    int k = 2;
    int h_row_ptr[] = {0, 2, 5, 7, 9, 10};
    int h_col_ind[] = {1, 4, 0, 2, 4, 1, 3, 2, 4, 3};
    int h_adj_list[] = {1, 4, 0, 2, 4, 1, 3, 2, 4, 3};
    int h_offset[] = {0, 2, 5, 7, 9, 10};
    int h_degrees[] = {2, 3, 2, 2, 3};
    
    computeKCore(h_row_ptr, h_col_ind, h_adj_list, h_offset, h_degrees, num_nodes, k);
    for (int i = 0; i < num_nodes; i++) {
        printf("Node %d -> Degree %d\n", i, h_degrees[i]);
    }
    

    
    return 0;
}
